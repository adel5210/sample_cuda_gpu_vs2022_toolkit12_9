#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <array>
#include <vector>
#include <random>
#include <cstdlib>
#include <ctime>

#define N 1000000

__global__ void add(int* a, int* b) {
	int index = threadIdx.x;
	int stride = blockDim.x;

	for (int i = index; i < N; i+=stride) {
		a[i] += b[i];
	}
}

int main()
{
	printf("Run CUDA samples\n");

	std::srand(static_cast<unsigned>(std::time(nullptr)));

	int* a;
	int* b;

	//allocate unified memory
	hipMallocManaged(&a, N * sizeof(int));
	hipMallocManaged(&b, N * sizeof(int));

	for (size_t i = 0; i < N; i++)
	{
		a[i] = std::rand() % 100;
		b[i] = std::rand() % 100;
	}

	add << <1, 256 >> > (a, b);

	//Block until the kernel is done
	hipDeviceSynchronize();

	// Free the unified memory
	hipFree(a);
	hipFree(b);

	return 0;
}